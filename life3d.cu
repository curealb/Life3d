#include "hip/hip_runtime.h"
/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24229109
 * 姓名: 于慧木
 * 邮箱: curealb@gmail.com
 ------------------------------------------------*/

 #include <chrono>
 #include <thread>
 #include <cstring>
 #include <fstream>
 #include <iostream>
 #include <string>
 #include <hip/hip_runtime.h>
 #include <>
 #include <omp.h>
 #include "hip/hip_runtime_api.h"
 
 #define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
 #define BLOCK_SIZE 4
 
 using std::cin, std::cout, std::endl;
 using std::ifstream, std::ofstream;
 
 // 检查是否算错
 bool check_universe(char* a,char* b,int N){
 
     for (int i = 0;i < N * N * N;i++) {
         if (a[i] != b[i])return false;
     }
 
     return true;
 }
 __global__ void print_test(){
     printf("Hello world from block %d, thread %d\n", blockIdx.x, threadIdx.x);
 }
 
 // 存活细胞数
 int population(int N, char *universe)
 {
     int result = 0;
     for (int i = 0; i < N * N * N; i++)
         result += universe[i];
     return result;
 }
 
 // 打印世界状态
 void print_universe(int N, char *universe)
 {
     // 仅在N较小(<= 32)时用于Debug
     if (N > 32)
         return;
     for (int x = 0; x < N; x++)
     {
         for (int y = 0; y < N; y++)
         {
             for (int z = 0; z < N; z++)
             {
                 if (AT(x, y, z))
                     cout << "O ";
                 else
                     cout << "* ";
             }
             cout << endl;
         }
         cout << endl;
     }
     cout << "population: " << population(N, universe) << endl;
 }
 
 __global__ void update_kernel(char* universe, char* next, int N) {
     int x = blockIdx.x * blockDim.x + threadIdx.x;
     int y = blockIdx.y * blockDim.y + threadIdx.y;
     int z = blockIdx.z * blockDim.z + threadIdx.z;
     printf("%d %d %d\t", x,y,z);
     // if (x >= N || y >= N || z >= N)return;
 
     // // 计算邻居的存活个数
     // int alive = 0;
     // for (int dx = -1;dx <= 1;dx++) {
     //     for (int dy = -1;dy <= 1;dy++) {
     //         for (int dz = -1;dz <= 1;dz++) {
     //             if (dx == 0 && dy == 0 && dz == 0)continue;
     //             int nx = (x + dx + N) % N;
     //             int ny = (y + dy + N) % N;
     //             int nz = (z + dz + N) % N;
     //             alive += AT(nx, ny, nz);
     //         }
     //     }
     // }
     
     // // 判断更新方式
     // if (AT(x, y, z) && (alive < 5 || alive > 7)) {
     //     next[x * N * N + y * N + z] = 0;
     // }
     // else if ((!AT(x, y, z) && alive == 6)) {
     //     next[x * N * N + y * N + z] = 1;
     // }
     // else {
     //     next[x * N * N + y * N + z] = AT(x, y, z);
     // }
 }
 
 void life3d_run_cuda(int N, char* universe, int T) {
     size_t size = N * N * N * sizeof(char);
 
     char* device_universe, * device_next;
 
     hipMalloc((void**)&device_universe, size);
     hipMalloc((void**)&device_next, size);
 
     hipMemcpy(device_universe, universe, size, hipMemcpyHostToDevice);
 
     dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
     dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y, (N + block.z - 1) / block.z);
 
     for (int t = 0;t < T;t++) {
         
         update_kernel << <grid,block >> > (device_universe, device_next, N);
         hipDeviceSynchronize();
 
 
         // 交换指向地址
         char* temp = device_universe;
         device_universe = device_next;
         device_next = temp;
     }
 
     hipMemcpy(universe, device_universe, size, hipMemcpyDeviceToHost);
 
     hipFree(device_universe);
     hipFree(device_next);
 }
 
 // 核心计算代码，将世界向前推进T个时刻
 void life3d_run(int N, char *universe, int T)
 {
     char *next = (char *)malloc(N * N * N);
     for (int t = 0; t < T; t++)
     {
         // outerloop: iter universe
         for (int x = 0; x < N; x++)
             for (int y = 0; y < N; y++)
                 for (int z = 0; z < N; z++)
                 {
                     // inner loop: stencil
                     int alive = 0;
                     for (int dx = -1; dx <= 1; dx++)
                         for (int dy = -1; dy <= 1; dy++)
                             for (int dz = -1; dz <= 1; dz++)
                             {
                                 if (dx == 0 && dy == 0 && dz == 0)
                                     continue;
                                 int nx = (x + dx + N) % N;
                                 int ny = (y + dy + N) % N;
                                 int nz = (z + dz + N) % N;
                                 alive += AT(nx, ny, nz);
                             }
                     if (AT(x, y, z) && (alive < 5 || alive > 7))
                         next[x * N * N + y * N + z] = 0;
                     else if (!AT(x, y, z) && alive == 6)
                         next[x * N * N + y * N + z] = 1;
                     else
                         next[x * N * N + y * N + z] = AT(x, y, z);
                 }
         memcpy(universe, next, N * N * N);
     }
     free(next); 
 }
 
 // 读取输入文件
 void read_file(char *input_file, char *buffer)
 {
     ifstream file(input_file, std::ios::binary | std::ios::ate);
     if (!file.is_open())
     {
         cout << "Error: Could not open file " << input_file << std::endl;
         exit(1);
     }
     std::streamsize file_size = file.tellg();
     file.seekg(0, std::ios::beg);
     if (!file.read(buffer, file_size))
     {
         std::cerr << "Error: Could not read file " << input_file << std::endl;
         exit(1);
     }
     file.close();
 }
 
 // 写入输出文件
 void write_file(char *output_file, char *buffer, int N)
 {
     ofstream file(output_file, std::ios::binary | std::ios::trunc);
     if (!file)
     {
         cout << "Error: Could not open file " << output_file << std::endl;
         exit(1);
     }
     file.write(buffer, N * N * N);
     file.close();
 }
 
 int main(int argc, char **argv)
 {
     // cmd args
     if (argc < 5)
     {
         cout << "usage: ./life3d N T input output" << endl;
         return 1;
     }
     int N = std::stoi(argv[1]);
     int T = std::stoi(argv[2]);
     char *input_file = argv[3];
     char *output_file = argv[4];
 
     char* universe = (char*)malloc(N * N * N);
     char *universe1 = (char *)malloc(N * N * N);
     read_file(input_file, universe);
     read_file(input_file, universe1);
 
     cout << "开始执行串行程序:" << endl;
     int start_pop = population(N, universe);
     auto start_time = std::chrono::high_resolution_clock::now();
     life3d_run(N, universe, T);
     auto end_time = std::chrono::high_resolution_clock::now();
     std::chrono::duration<double> duration = end_time - start_time;
     int final_pop = population(N, universe);
     write_file(output_file, universe, N);
 
     cout << "start population: " << start_pop << endl;
     cout << "final population: " << final_pop << endl;
     double time = duration.count();
     cout << "time: " << time << "s" << endl;
     cout << "cell per sec: " << T / time * N * N * N << endl;
 
 //------------------------------ --------------------------------//
     cout << "开始执行cuda程序:" << endl;
     start_pop = population(N, universe1);
     start_time = std::chrono::high_resolution_clock::now();
     life3d_run_cuda(N, universe1, T);
     end_time = std::chrono::high_resolution_clock::now();
     duration = end_time - start_time;
     final_pop = population(N, universe1);
     if (check_universe(universe, universe1, N)) {
         cout << "结果相同" << endl;
     }
     else {
         cout << "结果不同" << endl;
     }
 
 
     cout << "start population: " << start_pop << endl;
     cout << "final population: " << final_pop << endl;
     time = duration.count();
     cout << "time: " << time << "s" << endl;
     cout << "cell per sec: " << T / time * N * N * N << endl;
 
     
 
     free(universe);
     free(universe1);
     return 0;
 }
 